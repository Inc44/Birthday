// nvcc -o birthday birthday.cu -O3 -arch=sm_75 && ./birthday for GTX 1660 SUPER
// nvcc -o birthday birthday.cu -O3 -arch=sm_89 && ./birthday for RTX 4060 TI
#define BLOCK_SIZE 32
#define DAYS_IN_YEAR 365
#define NUM_BLOCKS (NUM_THREADS / BLOCK_SIZE)
#define NUM_THREADS 768	 // for GTX 1660 SUPER
// #define NUM_THREADS 2176 for RTX 4060 TI
#define PEOPLE 24
#define TOTAL_SIMULATIONS 1000000
#define MULTIPLIER 1664525
#define INCREMENT 1013904223
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
__global__ void simulate(int simulations,
						 int* d_successCount,
						 unsigned int seed) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int simulationsPerThread = simulations / NUM_THREADS;
	int successCount = 0;
	unsigned int state = seed ^ tid;
	for (int sim = 0; sim < simulationsPerThread; sim++) {
		int birthdays[DAYS_IN_YEAR] = {0};
		for (int i = 0; i < PEOPLE; i++) {
			state = state * MULTIPLIER + INCREMENT;
			int birthday = state % DAYS_IN_YEAR;
			birthdays[birthday]++;
		}
		int exactlyTwoCount = 0;
		for (int i = 0; i < DAYS_IN_YEAR; i++) {
			if (birthdays[i] == 2) {
				exactlyTwoCount++;
			}
		}
		if (exactlyTwoCount == 1) {
			successCount++;
		}
	}
	d_successCount[tid] = successCount;
}
int main() {
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);
	struct timespec time;
	clock_gettime(CLOCK_MONOTONIC, &time);
	unsigned long long seed = time.tv_sec * 1e9 + time.tv_nsec;
	int totalSimulations = TOTAL_SIMULATIONS;
	int *d_successCount, *h_successCount;
	hipMalloc((void**)&d_successCount, NUM_THREADS * sizeof(int));
	h_successCount = (int*)malloc(NUM_THREADS * sizeof(int));
	simulate<<<NUM_BLOCKS, BLOCK_SIZE>>>(totalSimulations, d_successCount,
										 seed);
	hipMemcpy(h_successCount, d_successCount, NUM_THREADS * sizeof(int),
			   hipMemcpyDeviceToHost);
	int totalSuccessCount = 0;
	for (int t = 0; t < NUM_THREADS; t++) {
		totalSuccessCount += h_successCount[t];
	}
	double probability = (double)totalSuccessCount / totalSimulations;
	printf("Probability: %.9f\n", probability);
	clock_gettime(CLOCK_MONOTONIC, &end);
	double elapsed =
		(end.tv_sec - start.tv_sec) + 1e-9 * (end.tv_nsec - start.tv_nsec);
	printf("Execution Time: %.3f s\n", elapsed);
	return 0;
}