// nvcc -o birthday birthday.cu -O3 -arch=sm_75 && ./birthday for GTX 1660 SUPER
// nvcc -o birthday birthday.cu -O3 -arch=sm_89 && ./birthday for RTX 4060 TI
#define BLOCK_SIZE 32
#define DAYS_IN_YEAR 365
#define NUM_BLOCKS (NUM_THREADS / BLOCK_SIZE)
#define NUM_THREADS 768	 // for GTX 1660 SUPER
// #define NUM_THREADS 2176 for RTX 4060 TI
#define PEOPLE 24
#define TOTAL_SIMULATIONS 1000000
#define MULTIPLIER 1664525
#define INCREMENT 1013904223
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
__global__ void simulate(int simulations,
						 int* d_successCount,
						 unsigned int seed) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int simulationsPerThread = simulations / NUM_THREADS;
	int localSuccessCount = 0;
	unsigned int state = seed ^ tid;
	for (int sim = 0; sim < simulationsPerThread; sim++) {
		int birthdays[DAYS_IN_YEAR] = {0};
		for (int i = 0; i < PEOPLE; i++) {
			state = state * MULTIPLIER + INCREMENT;
			int birthday = state % DAYS_IN_YEAR;
			birthdays[birthday]++;
		}
		int exactlyTwoCount = 0;
		for (int i = 0; i < DAYS_IN_YEAR; i++) {
			if (birthdays[i] == 2) {
				exactlyTwoCount++;
			}
		}
		if (exactlyTwoCount == 1) {
			localSuccessCount++;
		}
	}
	d_successCount[tid] = localSuccessCount;
}
int main() {
	struct timespec start_time, end_time;
	clock_gettime(CLOCK_MONOTONIC, &start_time);
	struct timespec time;
	clock_gettime(CLOCK_MONOTONIC, &time);
	unsigned long long seed = time.tv_sec * 1e9 + time.tv_nsec;
	int totalSimulations = TOTAL_SIMULATIONS;
	int *d_successCount, *h_successCount;
	hipMalloc((void**)&d_successCount, NUM_THREADS * sizeof(int));
	h_successCount = (int*)malloc(NUM_THREADS * sizeof(int));
	simulate<<<NUM_BLOCKS, BLOCK_SIZE>>>(totalSimulations, d_successCount,
										 seed);
	hipMemcpy(h_successCount, d_successCount, NUM_THREADS * sizeof(int),
			   hipMemcpyDeviceToHost);
	int totalSuccessCount = 0;
	for (int t = 0; t < NUM_THREADS; t++) {
		totalSuccessCount += h_successCount[t];
	}
	double probability = (double)totalSuccessCount / totalSimulations;
	printf("Probability: %.9f\n", probability);
	clock_gettime(CLOCK_MONOTONIC, &end_time);
	double elapsed_time =
		(end_time.tv_sec - start_time.tv_sec) + 1e-9 * (end_time.tv_nsec - start_time.tv_nsec);
	printf("Execution Time: %.3f s\n", elapsed_time);
	return 0;
}