#include "hip/hip_runtime.h"
// nvcc -o birthday birthday.cu -O3 -arch=sm_75 && ./birthday
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>

#include <stdlib.h>

#include <time.h>

#define BLOCK_SIZE 32
#define DAYS_IN_YEAR 365
#define NUM_BLOCKS (NUM_THREADS / BLOCK_SIZE)
#define NUM_THREADS 768
#define PEOPLE 24
#define TOTAL_SIMULATIONS 1000000
__global__ void simulate(int simulations, int * d_successCount, unsigned int currentTime) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int simulationsPerThread = simulations / NUM_THREADS;
  int successCount = 0;
  hiprandState state;
  hiprand_init(currentTime ^ tid, tid, 0, & state);
  for (int sim = 0; sim < simulationsPerThread; sim++) {
    int birthdays[DAYS_IN_YEAR] = {
      0
    };
    for (int i = 0; i < PEOPLE; i++) {
      int birthday = hiprand( & state) % DAYS_IN_YEAR;
      birthdays[birthday]++;
    }
    int exactlyTwoCount = 0;
    for (int i = 0; i < DAYS_IN_YEAR; i++) {
      if (birthdays[i] == 2) {
        exactlyTwoCount++;
      }
    }
    if (exactlyTwoCount == 1) {
      successCount++;
    }
  }
  d_successCount[tid] = successCount;
}
int main() {
  unsigned int seed = time(NULL);
  struct timespec start, end;
  clock_gettime(CLOCK_MONOTONIC, & start);
  int totalSimulations = TOTAL_SIMULATIONS;
  int * d_successCount, * h_successCount;
  hipMalloc((void ** ) & d_successCount, NUM_THREADS * sizeof(int));
  h_successCount = (int * ) malloc(NUM_THREADS * sizeof(int));
  simulate << < NUM_BLOCKS, BLOCK_SIZE >>> (totalSimulations, d_successCount, seed);
  hipMemcpy(h_successCount, d_successCount, NUM_THREADS * sizeof(int), hipMemcpyDeviceToHost);
  int totalSuccessCount = 0;
  for (int t = 0; t < NUM_THREADS; t++) {
    totalSuccessCount += h_successCount[t];
  }
  double probability = (double) totalSuccessCount / totalSimulations;
  printf("Probability: %.9f\n", probability);
  clock_gettime(CLOCK_MONOTONIC, & end);
  double elapsed = (end.tv_sec - start.tv_sec) + 1e-9 * (end.tv_nsec - start.tv_nsec);
  printf("Execution Time: %.3f s\n", elapsed);
  return 0;
}