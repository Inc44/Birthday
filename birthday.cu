// nvcc -o birthday birthday.cu -O3 -arch=sm_75 && ./birthday for GTX 1660 SUPER
// nvcc -o birthday birthday.cu -O3 -arch=sm_89 && ./birthday for RTX 4060 TI
#define BLOCK_SIZE 32
#define DAYS_IN_YEAR 365
#define NUM_BLOCKS (NUM_THREADS / BLOCK_SIZE)
#define NUM_THREADS 768	 // for GTX 1660 SUPER
// #define NUM_THREADS 2176 for RTX 4060 TI
#define PEOPLE 24
#define TOTAL_SIMULATIONS 1000000
#define MULTIPLIER 1664525
#define INCREMENT 1013904223
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>
__global__ void simulate(uint32_t simulations,
						 uint32_t* deviceSuccessCount,
						 uint64_t seed) {
	uint16_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t simulationsPerThread = simulations / NUM_THREADS;
	uint32_t localSuccessCount = 0;
	uint32_t state = seed ^ threadId;
	for (uint32_t sim = 0; sim < simulationsPerThread; sim++) {
		uint8_t birthdays[DAYS_IN_YEAR] = {0};
		for (uint8_t i = 0; i < PEOPLE; i++) {
			state = state * MULTIPLIER + INCREMENT;
			uint16_t birthday = state % DAYS_IN_YEAR;
			birthdays[birthday]++;
		}
		uint8_t exactlyTwoCount = 0;
		for (uint16_t i = 0; i < DAYS_IN_YEAR; i++) {
			if (birthdays[i] == 2) {
				exactlyTwoCount++;
			}
		}
		if (exactlyTwoCount == 1) {
			localSuccessCount++;
		}
	}
	deviceSuccessCount[threadId] = localSuccessCount;
}
int main() {
	struct timespec start_time, end_time;
	clock_gettime(CLOCK_MONOTONIC, &start_time);
	struct timespec time;
	clock_gettime(CLOCK_MONOTONIC, &time);
	uint64_t seed = time.tv_sec * 1e9 + time.tv_nsec;
	uint32_t *deviceSuccessCount, *hostSuccessCount;
	hipMalloc((void**)&deviceSuccessCount, NUM_THREADS * sizeof(uint32_t));
	hostSuccessCount = (uint32_t*)malloc(NUM_THREADS * sizeof(uint32_t));
	simulate<<<NUM_BLOCKS, BLOCK_SIZE>>>(TOTAL_SIMULATIONS, deviceSuccessCount,
										 seed);
	hipMemcpy(hostSuccessCount, deviceSuccessCount, NUM_THREADS * sizeof(uint32_t),
			   hipMemcpyDeviceToHost);
	uint32_t totalSuccessCount = 0;
	for (uint16_t t = 0; t < NUM_THREADS; t++) {
		totalSuccessCount += hostSuccessCount[t];
	}
	double probability = (double)totalSuccessCount / TOTAL_SIMULATIONS;
	printf("Probability: %.9f\n", probability);
	clock_gettime(CLOCK_MONOTONIC, &end_time);
	double elapsed_time =
		(end_time.tv_sec - start_time.tv_sec) + 1e-9 * (end_time.tv_nsec - start_time.tv_nsec);
	printf("Execution Time: %.3f s\n", elapsed_time);
	return 0;
}