// nvcc -o birthday birthday.cu -O3 -arch=sm_75 && ./birthday for GTX 1660 SUPER
// nvcc -o birthday birthday.cu -O3 -arch=sm_89 && ./birthday for RTX 4060 TI
#define BLOCK_SIZE 32
#define DAYS_IN_YEAR 365
#define NUM_BLOCKS (NUM_THREADS / BLOCK_SIZE)
#define NUM_THREADS 768	 // for GTX 1660 SUPER
// #define NUM_THREADS 2176 for RTX 4060 TI
#define PEOPLE 24
#define TOTAL_SIMULATIONS 1000000
#define MULTIPLIER 1664525
#define INCREMENT 1013904223
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>
__global__ void simulate(uint32_t simulations,
						 uint32_t* d_successCount,
						 uint64_t seed) {
	uint16_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t simulationsPerThread = simulations / NUM_THREADS;
	uint32_t localSuccessCount = 0;
	uint32_t state = seed ^ tid;
	for (uint32_t sim = 0; sim < simulationsPerThread; sim++) {
		uint8_t birthdays[DAYS_IN_YEAR] = {0};
		for (uint8_t i = 0; i < PEOPLE; i++) {
			state = state * MULTIPLIER + INCREMENT;
			uint16_t birthday = state % DAYS_IN_YEAR;
			birthdays[birthday]++;
		}
		uint8_t exactlyTwoCount = 0;
		for (uint16_t i = 0; i < DAYS_IN_YEAR; i++) {
			if (birthdays[i] == 2) {
				exactlyTwoCount++;
			}
		}
		if (exactlyTwoCount == 1) {
			localSuccessCount++;
		}
	}
	d_successCount[tid] = localSuccessCount;
}
int main() {
	struct timespec start_time, end_time;
	clock_gettime(CLOCK_MONOTONIC, &start_time);
	struct timespec time;
	clock_gettime(CLOCK_MONOTONIC, &time);
	uint64_t seed = time.tv_sec * 1e9 + time.tv_nsec;
	uint32_t *d_successCount, *h_successCount;
	hipMalloc((void**)&d_successCount, NUM_THREADS * sizeof(uint32_t));
	h_successCount = (uint32_t*)malloc(NUM_THREADS * sizeof(uint32_t));
	simulate<<<NUM_BLOCKS, BLOCK_SIZE>>>(TOTAL_SIMULATIONS, d_successCount,
										 seed);
	hipMemcpy(h_successCount, d_successCount, NUM_THREADS * sizeof(uint32_t),
			   hipMemcpyDeviceToHost);
	uint32_t totalSuccessCount = 0;
	for (uint16_t t = 0; t < NUM_THREADS; t++) {
		totalSuccessCount += h_successCount[t];
	}
	double probability = (double)totalSuccessCount / TOTAL_SIMULATIONS;
	printf("Probability: %.9f\n", probability);
	clock_gettime(CLOCK_MONOTONIC, &end_time);
	double elapsed_time =
		(end_time.tv_sec - start_time.tv_sec) + 1e-9 * (end_time.tv_nsec - start_time.tv_nsec);
	printf("Execution Time: %.3f s\n", elapsed_time);
	return 0;
}